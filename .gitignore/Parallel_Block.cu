
#include "hip/hip_runtime.h"

#include <stdio.h>
#define N (33 * 1024)

__global__ void add(int *a, int *b, int * c)
{
	//threadIdx.x:当前线程的Index. blockIdx:当前线程块的index. blockDim.x:每个线程块中线程的数量.
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while(tid < N)
	{
		c[tid] = a[tid] + b[tid];
		//blockDim.x:每个线程块中线程的数量. gridDim.x:线程格中线程块的数量.
		tid += blockDim.x * gridDim.x;
	}
}
int main()
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a, N*sizeof(int));
	hipMalloc((void**)&dev_b, N*sizeof(int));
	hipMalloc((void**)&dev_c, N*sizeof(int));
	for(int i = 0; i < N; ++i)
	{
		a[i] = -i;
		b[i] = i*i;
	}
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);
	//add<<<a, b>>>其中a表示设备在执行核函数时使用的并行线程块的数量，b表示一个线程块中有b个线程.(其中b不能超过512)
	add<<<128, 128>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	bool success = true;
	for(int i = 0; i < N; ++i)
	{
		if((a[i] + b[i]) != c[i]){
			printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
	}
	if(success)
		printf("We did it!\n");
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
