#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "String.h"
#define N 10

__global__ void add(int *a, int *b, int * c)
{
	int tid = blockIdx.x;
	if(tid < N)
		c[tid] = a[tid] + b[tid];
}
int main()
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a, N*sizeof(int));
	hipMalloc((void**)&dev_b, N*sizeof(int));
	hipMalloc((void**)&dev_c, N*sizeof(int));
	for(int i = 0; i < N; ++i)
	{
		a[i] = -i;
		b[i] = i*i;
	}
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);
	//N:表示设备在执行核函数时使用的并行线程块的数量，
	add<<<N, 1>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i < N; ++i)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	//Second Demo
#if 0
	int count = 0;
	hipDeviceProp_t prop;
	memset(&prop, 0x00, sizeof(hipDeviceProp_t));
	hipGetDeviceCount(&count);
	printf("Device Count is %d\n", count);
	for(int i = 0; i <count; ++i)
	{
		hipGetDeviceProperties(&prop, i);
		printf("Information for Device %d\n", i);
		printf("Name:%s\n", prop.name);
	}
#endif
	//Third Demo. 多GPU环境下选择最优的GPU.
	hipDeviceProp_t prop;
	int dev;
	hipGetDevice(&dev);
	printf("ID of current CUDA device: %d\n", dev);
	memset(&prop, 0x00, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 3;
	hipChooseDevice(&dev, &prop);
	printf("ID of CUDA device closest to revision 1.3: %d\n", dev);
	hipSetDevice(dev);
    return 0;
}
